#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <unistd.h>

#define GIG 1073741824

__device__ char d_data[GIG];
__device__ char *d_c;

float sum;
int i;
char h_data[GIG];
char *h_c;

main(){
  clock_t start, end;
  double elapsed;

  start = clock();

printf("transferring 1GB back and forth...\n");
  for (i=0;i<1;i++){
    hipMemcpyToSymbol(HIP_SYMBOL(d_data), h_data, GIG, 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(h_data), d_data, GIG, 0, hipMemcpyDeviceToHost);
  }
  end = clock();
  elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;
  printf("time elapsed is %f.\n\n", elapsed);

  start = clock();

printf("transferring 1B back and forth...\n");
  for (i=0;i<100000;i++){
    hipMemcpyToSymbol(HIP_SYMBOL(d_c), h_c, 1, 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(h_c), d_c, 1, 0, hipMemcpyDeviceToHost);
  }
  end = clock();
  elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;
  printf("time elapsed is %f.\n\n", elapsed);

  
  
}
