#include "hip/hip_runtime.h"
/* CUDAEnergy.cu */

/*****************************************************************************/
/*                                                                           */
/*                                                                           */
/*****************************************************************************/

#include <genels.h>

extern "C" void calculateEnergy(struct ZThread *h_thread);
extern "C" void setResolution(float *gross_resolution, float *fine_resolution);

__device__ float _gross_resolution;
__device__ float _fine_resolution;

__global__ void EnergyKernel(struct ZThread *d_thread, struct EnergyArray *d_energy_array)
{
  unsigned int idx = blockIdx.x;
  unsigned int idy = blockIdx.y;
  unsigned int idz = threadIdx.x;

  float repulsion=0;
  float attraction=0;
  float alpha;
  float dx, dy, dz, d, dd;

  float cuda_x = d_thread->i * _gross_resolution + idx * _fine_resolution;
  float cuda_y = d_thread->j * _gross_resolution + idy * _fine_resolution;
  float cuda_z = d_thread->k * _gross_resolution + idz * _fine_resolution;

  // evaluate energy at (cuda_x, cuda_y, cuda_z);
  for (int i=0; i< d_thread->verlet.close_atoms; i++)
  {
    // central atom
    dx = d_thread->verlet.x[i] - cuda_x;
    dy = d_thread->verlet.y[i] - cuda_y;
    dz = d_thread->verlet.z[i] - cuda_z;
    dd = dx*dx + dy*dy + dz*dz;
    d = sqrt(dd);
    alpha = pow(d_thread->verlet.r_ij[i], 3) / (d * dd);   
    repulsion += d_thread->verlet.epsilon_ij[i] * alpha * alpha * alpha;
    attraction += d_thread->verlet.epsilon_ij[i] * alpha * alpha;
  } 

  d_energy_array->energy[idx][idy][idz] = (2 * repulsion - 3 * attraction);
}

extern "C" void setResolution(float *gross_resolution, float *fine_resolution)
{
  hipMemcpyToSymbol((const char*)&_gross_resolution, gross_resolution, sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol((const char*)&_fine_resolution, fine_resolution, sizeof(float), 0, hipMemcpyHostToDevice);
}

extern "C" void calculateEnergy(struct ZThread *thread)
{
  hipError_t err;
  struct ZThread *d_thread; // pointer for device memory
  struct EnergyArray *d_energy_array;
  size_t TmemSize = sizeof(struct ZThread);

  err = hipErrorUnknown;
  while (err != hipSuccess)
  {
    hipMalloc( (void **) &d_thread, TmemSize );
    err = hipGetLastError();
    if (err != hipSuccess) {fprintf(stderr, "%s\n", hipGetErrorString(err)); sleep(1);}
  }

  err = hipErrorUnknown;
  while (err != hipSuccess)
  {
    hipMalloc( (void **) &d_energy_array, sizeof(struct EnergyArray) );
    err = hipGetLastError();
    if (err != hipSuccess) {fprintf(stderr, "%s\n", hipGetErrorString(err)); sleep(1);}
  }

  dim3 dimGrid(FINE_GRID_RESOLUTION, FINE_GRID_RESOLUTION);
  dim3 dimBlock(FINE_GRID_RESOLUTION, 1, 1);

  err = hipErrorUnknown;
  while (err != hipSuccess)
  {
    hipMemcpy( d_thread, thread, TmemSize, hipMemcpyHostToDevice );
    err = hipGetLastError();
    if (err != hipSuccess) {fprintf(stderr, "%s\n", hipGetErrorString(err)); sleep(1);}
  }

  EnergyKernel<<< dimGrid, dimBlock >>>( d_thread, d_energy_array );
  hipDeviceSynchronize(); // block until the device has completed
  err = hipGetLastError();
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err)); 

  hipMemcpy( thread, d_thread, TmemSize, hipMemcpyDeviceToHost );
  err = hipGetLastError();
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err)); 

  hipMemcpy( thread->energy_array, d_energy_array, sizeof(struct EnergyArray), hipMemcpyDeviceToHost );
  err = hipGetLastError();
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err)); 

  // free device memory
  hipFree(d_thread);
  hipFree(d_energy_array);
  return;
}

