// CopyBackAndForth.cu


#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>

__device__ char devarray[16];

extern "C" void runTest()
{
   char zerobuf[16];
   memset(zerobuf, '@', sizeof(zerobuf));

   hipError_t r = hipMemcpyToSymbol(HIP_SYMBOL(devarray), zerobuf, sizeof(zerobuf), 0, hipMemcpyHostToDevice);
   assert(hipSuccess == r);

   char out[16];
   r = hipMemcpyFromSymbol(out, HIP_SYMBOL(devarray), sizeof(devarray), 0, hipMemcpyDeviceToHost);
   assert(hipSuccess == r);

   assert(memcmp(out, zerobuf, sizeof(out)) == 0);
}

int main()
{
  runTest();
}
