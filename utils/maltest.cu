/* maltest.cu */

#include <ftw_config_parser.h>

#include <ftw_types.h>
#include <stdio.h>
#include <math.h>
#include <unistd.h>

main(int argc, char *argv[]) 
{
  int device_count;
  hipError_t err = hipErrorUnknown;

//  REMOVING either device count or the readGFG ALLOWS THE MALLOC TO WORK.  WTF?
  hipGetDeviceCount(&device_count);
  printf("%d device found.\n", device_count);

err=hipGetLastError();
fprintf(stderr, "main::%s\n", hipGetErrorString(err));

  ftw_GFG65536 *gfg = readGFG65536(stdin);


  int *p_int;

fprintf(stderr, "main::%s\n", hipGetErrorString(err));
fprintf(stderr, "main::p_int:  %p\n", p_int);
err = hipMalloc( (void **) &p_int, 65536 * sizeof(int));
fprintf(stderr, "main::p_int:  %p\n", p_int);
fprintf(stderr, "main::%s\n", hipGetErrorString(err));

sleep(1);

hipSetDevice(1);

err = hipMalloc( (void **) &p_int, 65536 * sizeof(int));
fprintf(stderr, "main::%s\n", hipGetErrorString(err));
}


