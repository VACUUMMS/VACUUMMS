#include "hip/hip_runtime.h"
/* ftw_gfg2fvi.cu */

//  IN:    A pointer to a ***non-replicated*** polymer configuration as *ftw_GFG65536.  
//  OUT:   A free volume intensity is returned as *ftw_FVI256.  
//  Input configuration is not modified.  
//  Cross-interaction values are stored for the replicated config only.

#include <ftw_gfg2fvi.h>
#include <ftw_config_parser.h>
#include <ftw_types.h>

#include <stdlib.h>
#include <math.h>

// I took the kernel prototype out of the header file, because the header is included by C/C++ compilers that don't know what a kernel is...
// NOTE:  this uses COMPASS / LJ 6-9 potential
__global__ void EnergyKernel256(ftw_GFG65536 *d_configuration, ftw_EnergyArray256 *d_attraction, ftw_EnergyArray256 *d_repulsion, ftw_EnergyArray256 *d_total) 
{
  unsigned int idx = blockIdx.x;
  unsigned int idy = blockIdx.y;
  unsigned int idz = threadIdx.x;

  float repulsion=0;
  float attraction=0;
  float alpha;
  float dx, dy, dz, d, dd;
  float f_resolution_x = d_configuration->box_x / 256;
  float f_resolution_y = d_configuration->box_y / 256;
  float f_resolution_z = d_configuration->box_z / 256;

  float cuda_x = idx * f_resolution_x;
  float cuda_y = idy * f_resolution_y;
  float cuda_z = idz * f_resolution_z;

  // evaluate energy at (cuda_x, cuda_y, cuda_z);
  for (int i=0; i< d_configuration->n_atoms; i++) {
    // central atom
    dx = d_configuration->atom[i].x - cuda_x;
    dy = d_configuration->atom[i].y - cuda_y;
    dz = d_configuration->atom[i].z - cuda_z;
    dd = dx*dx + dy*dy + dz*dz; d = sqrt(dd);
    alpha = pow(d_configuration->atom[i].sigma, 3) / (d * dd);   
    repulsion += d_configuration->atom[i].epsilon * alpha * alpha * alpha;
    attraction += d_configuration->atom[i].epsilon * alpha * alpha;
  } 

  // If NULL pointers are passed for the attraction or repulsion, no values are returned.
  if (d_attraction) d_attraction->energy[idx][idy][idz] = 3 * attraction;
  if (d_repulsion) d_repulsion->energy[idx][idy][idz] = 2 * repulsion;
  if (d_total) d_total->energy[idx][idy][idz] = 2 * repulsion - 3 * attraction;
}

__global__ void EnergyKernel256_612(ftw_GFG65536 *d_configuration, ftw_EnergyArray256 *d_attraction, ftw_EnergyArray256 *d_repulsion, ftw_EnergyArray256 *d_total) 
{
  unsigned int idx = blockIdx.x;
  unsigned int idy = blockIdx.y;
  unsigned int idz = threadIdx.x;

  float repulsion=0;
  float attraction=0;
  float sigma_over_r_sq;
  float dx, dy, dz, dd;
  float f_resolution_x = d_configuration->box_x / 256;
  float f_resolution_y = d_configuration->box_y / 256;
  float f_resolution_z = d_configuration->box_z / 256;

  float cuda_x = idx * f_resolution_x;
  float cuda_y = idy * f_resolution_y;
  float cuda_z = idz * f_resolution_z;

  // evaluate energy at (cuda_x, cuda_y, cuda_z);
  for (int i=0; i< d_configuration->n_atoms; i++) {
    // central atom
    dx = d_configuration->atom[i].x - cuda_x;
    dy = d_configuration->atom[i].y - cuda_y;
    dz = d_configuration->atom[i].z - cuda_z;
    dd = dx*dx + dy*dy + dz*dz; 
    sigma_over_r_sq = d_configuration->atom[i].sigma * d_configuration->atom[i].sigma / dd; // squared   
    repulsion += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
    attraction += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
  } 

  // If NULL pointers are passed for the attraction or repulsion, no values are returned.
  if (d_attraction) d_attraction->energy[idx][idy][idz] = 4 * attraction;
  if (d_repulsion) d_repulsion->energy[idx][idy][idz] = 4 * repulsion;
  if (d_total) d_total->energy[idx][idy][idz] = 4 * repulsion - 4 * attraction;
}

__global__ void EnergyKernel512_612(ftw_GFG65536 *d_configuration, ftw_EnergyArray512 *d_attraction, ftw_EnergyArray512 *d_repulsion, ftw_EnergyArray512 *d_total) 
{
  unsigned int idx = blockIdx.x;
  unsigned int idy = blockIdx.y;
  unsigned int idz = threadIdx.x;

  float repulsion=0;
  float attraction=0;
  float sigma_over_r_sq;
  float dx, dy, dz, dd;
  float f_resolution_x = d_configuration->box_x / 256;
  float f_resolution_y = d_configuration->box_y / 256;
  float f_resolution_z = d_configuration->box_z / 256;

  float cuda_x = idx * f_resolution_x;
  float cuda_y = idy * f_resolution_y;
  float cuda_z = idz * f_resolution_z;

  // evaluate energy at (cuda_x, cuda_y, cuda_z);
  for (int i=0; i< d_configuration->n_atoms; i++) {
    // central atom
    dx = d_configuration->atom[i].x - cuda_x;
    dy = d_configuration->atom[i].y - cuda_y;
    dz = d_configuration->atom[i].z - cuda_z;
    dd = dx*dx + dy*dy + dz*dz; 
    sigma_over_r_sq = d_configuration->atom[i].sigma * d_configuration->atom[i].sigma / dd; // squared   
    repulsion += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
    attraction += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
  } 

  // If NULL pointers are passed for the attraction or repulsion, no values are returned.
  if (d_attraction) d_attraction->energy[idx][idy][idz] = 4 * attraction;
  if (d_repulsion) d_repulsion->energy[idx][idy][idz] = 4 * repulsion;
  if (d_total) d_total->energy[idx][idy][idz] = 4 * repulsion - 4 * attraction;
}

//  This is the routine to call from outside the library
extern "C" ftw_FVI256 *GFGToFVI256(ftw_GFG65536 *gfg, float sigma, float epsilon) 
{
  // call energy array then process each val
  ftw_EnergyArray256 *era = GFGToRepulsion256(gfg, sigma, epsilon);
  ftw_FVI256 *fvi = (ftw_FVI256*)malloc(sizeof(ftw_FVI256));

  // now process each value...
  for (int i=0; i<256; i++) for (int j=0; j<256; j++) for (int k=0; k<256; k++)
    fvi->intensity[i][j][k] = exp(era->energy[i][j][k]/-298000); // this is arbitrary... should be clarified
  
  return fvi;
}

//  This routine to be called from outside the library
extern "C" ftw_EnergyArray256 *GFGToRepulsion256_612(ftw_GFG65536 *gfg, float sigma, float epsilon)
{
  ftw_EnergyArray256 	*d_repulsion;
  ftw_GFG65536 		*d_configuration;

fprintf(stderr, "pointers:  %ld\t%ld\n", d_repulsion, d_configuration);
fflush(stderr);

  // replicate the gfg
  ftw_GFG65536 *h_configuration = replicateGFG65536(gfg); 

// and cross-parameterize use 6-12 rule
  for (int n=0; n<gfg->n_atoms; n++)
  {
    h_configuration->atom[n].sigma = 0.5f * (sigma + h_configuration->atom[n].sigma);
    h_configuration->atom[n].epsilon = sqrt(epsilon * h_configuration->atom[n].epsilon);
  }

  // then do the calc
  hipError_t err;
  /* allocate for energy array and configuration on device */
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_repulsion, sizeof(ftw_EnergyArray256)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_configuration, sizeof(ftw_GFG65536)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy( d_configuration, h_configuration, sizeof(ftw_GFG65536), hipMemcpyHostToDevice ));

  dim3 dimGrid(256, 256);
  dim3 dimBlock(256, 1, 1);

  EnergyKernel256_612<<< dimGrid, dimBlock >>>(d_configuration, NULL, d_repulsion, NULL);
  hipDeviceSynchronize(); // block until the device has completed
  err = hipGetLastError();
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err)); 

  // retrieve result
  ftw_EnergyArray256 *h_repulsion = (ftw_EnergyArray256 *)malloc(sizeof(ftw_EnergyArray256));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy(h_repulsion, d_repulsion, sizeof(ftw_EnergyArray256), hipMemcpyDeviceToHost ));

  // free device memory
  hipFree(d_configuration);
  hipFree(d_repulsion);

  free(h_configuration); // free host memory for replicated configuration

  return h_repulsion;
}

//  This routine to be called from outside the library
extern "C" ftw_EnergyArray512 *GFGToRepulsion512_612(ftw_GFG65536 *gfg, float sigma, float epsilon)
{
  ftw_EnergyArray512 	*d_repulsion;
  ftw_GFG65536 		*d_configuration;


  // replicate the gfg
  ftw_GFG65536 *h_configuration = replicateGFG65536(gfg); 

// and cross-parameterize use 6-12 rule
  for (int n=0; n<gfg->n_atoms; n++)
  {
    h_configuration->atom[n].sigma = 0.5f * (sigma + h_configuration->atom[n].sigma);
    h_configuration->atom[n].epsilon = sqrt(epsilon * h_configuration->atom[n].epsilon);
  }

  // then do the calc
  hipError_t err;
  /* allocate for energy array and configuration on device */
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_repulsion, sizeof(ftw_EnergyArray256)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_configuration, sizeof(ftw_GFG65536)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy( d_configuration, h_configuration, sizeof(ftw_GFG65536), hipMemcpyHostToDevice ));

  dim3 dimGrid(512, 512);
  dim3 dimBlock(512, 1, 1);

  EnergyKernel512_612<<< dimGrid, dimBlock >>>(d_configuration, NULL, d_repulsion, NULL);
  hipDeviceSynchronize(); // block until the device has completed
  err = hipGetLastError();
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err)); 

  // retrieve result
  ftw_EnergyArray512 *h_repulsion = (ftw_EnergyArray512 *)malloc(sizeof(ftw_EnergyArray512));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy(h_repulsion, d_repulsion, sizeof(ftw_EnergyArray512), hipMemcpyDeviceToHost ));

  // free device memory
  hipFree(d_configuration);
  hipFree(d_repulsion);

  free(h_configuration); // free host memory for replicated configuration

  return h_repulsion;
}

//  This routine to be called from outside the library
extern "C" ftw_EnergyArray256 *GFGToRepulsion256(ftw_GFG65536 *gfg, float sigma, float epsilon)
{
  ftw_EnergyArray256 	*d_repulsion;
  ftw_GFG65536 		*d_configuration;

  // replicate the gfg
  ftw_GFG65536 *h_configuration = replicateGFG65536(gfg); 

  // and cross-parameterize 
  for (int n=0; n<gfg->n_atoms; n++)
  {
    h_configuration->atom[n].sigma = pow(0.5f * (float)(pow(sigma, 6) + pow(h_configuration->atom[n].sigma, 6)), 0.1666666f);
    h_configuration->atom[n].epsilon = sqrt(epsilon * h_configuration->atom[n].epsilon);
  }

  // then do the calc
  hipError_t err;
  /* allocate for energy array and configuration on device */
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_repulsion, sizeof(ftw_EnergyArray256)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_configuration, sizeof(ftw_GFG65536)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy( d_configuration, h_configuration, sizeof(ftw_GFG65536), hipMemcpyHostToDevice ));

  dim3 dimGrid(256, 256);
  dim3 dimBlock(256, 1, 1);

  EnergyKernel256<<< dimGrid, dimBlock >>>(d_configuration, NULL, d_repulsion, NULL);
  hipDeviceSynchronize(); // block until the device has completed
  err = hipGetLastError();
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err)); 

  // retrieve result
  ftw_EnergyArray256 *h_repulsion = (ftw_EnergyArray256 *)malloc(sizeof(ftw_EnergyArray256));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy(h_repulsion, d_repulsion, sizeof(ftw_EnergyArray256), hipMemcpyDeviceToHost ));

  // free device memory
  hipFree(d_configuration);
  hipFree(d_repulsion);

  free(h_configuration); // free host memory for replicated configuration

  return h_repulsion;
}

//  This routine to be called from outside the library
extern "C" ftw_EnergyArray256 *GFGToEnergyArray256(ftw_GFG65536 *gfg, float sigma, float epsilon)
{
  ftw_EnergyArray256 	*d_energy_array;
  ftw_GFG65536 		*d_configuration;

  // replicate the gfg
  ftw_GFG65536 *h_configuration = replicateGFG65536(gfg); 

  // and cross-parameterize 
  for (int n=0; n<gfg->n_atoms; n++)
  {
    h_configuration->atom[n].sigma = pow(0.5f * (float)(pow(sigma, 6) + pow(h_configuration->atom[n].sigma, 6)), 0.1666666f);
    h_configuration->atom[n].epsilon = sqrt(epsilon * h_configuration->atom[n].epsilon);
  }

  // then do the calc
  hipError_t err;
  /* allocate for energy array and configuration on device */
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_energy_array, sizeof(ftw_EnergyArray256)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_configuration, sizeof(ftw_GFG65536)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy( d_configuration, h_configuration, sizeof(ftw_GFG65536), hipMemcpyHostToDevice ));

  dim3 dimGrid(256, 256);
  dim3 dimBlock(256, 1, 1);

  EnergyKernel256<<< dimGrid, dimBlock >>>(d_configuration, NULL, NULL, d_energy_array);
  hipDeviceSynchronize(); // block until the device has completed
  err = hipGetLastError();
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err)); 

  // retrieve result
  ftw_EnergyArray256 *h_energy_array = (ftw_EnergyArray256 *)malloc(sizeof(ftw_EnergyArray256));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy(h_energy_array, d_energy_array, sizeof(ftw_EnergyArray256), hipMemcpyDeviceToHost ));

  // free device memory
  hipFree(d_configuration);
  hipFree(d_energy_array);

  free(h_configuration); // free host memory for replicated configuration

  return h_energy_array;
}

// This is for a traditional LJ 6-12 interaction.  Note that sigma is the value where energy is zero, not the well-bottom as for COMPASS...
// This operates on 'chunks' in x-direction because the domain is too large for the GPU memory
__global__ void EnergyKernel1024_612(	ftw_GFG65536 *d_configuration, 
 					ftw_Chunk *d_attraction, 
 					ftw_Chunk *d_repulsion, 
 					ftw_Chunk *d_total, 
 					int chunk, int chunk_size) {
  unsigned int idx = threadIdx.x;
  unsigned int idy = blockIdx.x;
  unsigned int idz = blockIdx.y;

  float repulsion=0;
  float attraction=0;
  float sigma_over_r_sq;
  float dx, dy, dz, dd;
  float f_resolution_x = d_configuration->box_x / 1024;
  float f_resolution_y = d_configuration->box_y / 1024;
  float f_resolution_z = d_configuration->box_z / 1024;

  float cuda_x = (chunk * chunk_size + idx ) * f_resolution_x;
  float cuda_y = idy * f_resolution_y;
  float cuda_z = idz * f_resolution_z;

  // evaluate energy at (cuda_x, cuda_y, cuda_z);
  for (int i=0; i< d_configuration->n_atoms; i++) {
    // central atom
    dx = d_configuration->atom[i].x - cuda_x;
    dy = d_configuration->atom[i].y - cuda_y;
    dz = d_configuration->atom[i].z - cuda_z;
    dd = dx*dx + dy*dy + dz*dz;
    sigma_over_r_sq = d_configuration->atom[i].sigma * d_configuration->atom[i].sigma / dd;
    repulsion  += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
    attraction += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
  } 

  // If NULL pointers are passed, then no values are recorded.
  if (d_attraction) d_attraction->energy[idx][idy][idz] = 4 * attraction;
  if (d_repulsion) d_repulsion->energy[idx][idy][idz] = 4 * repulsion;
  if (d_total) d_total->energy[idx][idy][idz] = 4 * repulsion - 4 * attraction;
}

// This routine to be called from outside the library
extern "C" ftw_EnergyArray1024 *GFGToEnergyArray1024_612(ftw_GFG65536 *gfg, float sigma, float epsilon)
{
  ftw_Chunk 		*d_energy_array_chunk;
  ftw_GFG65536 		*d_configuration;

  ftw_GFG65536 *h_configuration = replicateGFG65536(gfg); // replicate the gfg 
  for (int n=0; n<gfg->n_atoms; n++) // and cross-parameterize 
  {
    h_configuration->atom[n].sigma = pow(0.5f * (float)(pow(sigma, 6) + pow(h_configuration->atom[n].sigma, 6)), 0.1666666f);
    h_configuration->atom[n].epsilon = sqrt(epsilon * h_configuration->atom[n].epsilon);
  }

  // then do the calc
  // (x,y,z) is (blockx, gridx, gridy)... chunking to 4 parts in x, then will combine results
  int chunk_size = 256, chunks = 4;
  dim3 dimGrid(1024, 1024);
  dim3 dimBlock(chunk_size, 1, 1);

  hipError_t err;
  /* allocate for energy array and configuration on device */
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_energy_array_chunk, sizeof(ftw_Chunk)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_configuration, sizeof(ftw_GFG65536)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy( d_configuration, h_configuration, sizeof(ftw_GFG65536), hipMemcpyHostToDevice ));

  ftw_EnergyArray1024 *h_energy_array = (ftw_EnergyArray1024 *)malloc(sizeof(ftw_EnergyArray1024)); // host structure, for result

  for (int chunk=0; chunk < chunks; chunk++)
  {
    EnergyKernel1024_612<<< dimGrid, dimBlock >>>(d_configuration, NULL, NULL, d_energy_array_chunk, chunk, chunk_size);
    hipDeviceSynchronize(); // block until the device has completed
    err = hipGetLastError();
    if (err != hipSuccess) {printf("CUDA error:  %s\n", hipGetErrorString(err)); exit(1);}
    // retrieve result
    ftw_EnergyArray1024* h_address = (ftw_EnergyArray1024*)((long)h_energy_array + (long)(sizeof(ftw_Chunk) * chunk));
    for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy(h_address, d_energy_array_chunk, sizeof(ftw_Chunk), hipMemcpyDeviceToHost ));
  }

  // free device memory
  hipFree(d_configuration);
  hipFree(d_energy_array_chunk);

  free(h_configuration); // free host memory for replicated configuration
  return h_energy_array;
}

// This routine to be called from outside the library
extern "C" ftw_EnergyArray1024 *GFGToRepulsion1024_612(ftw_GFG65536 *gfg, float sigma, float epsilon)
{
  ftw_Chunk 		*d_repulsion_chunk;
  ftw_GFG65536 		*d_configuration;

  ftw_GFG65536 *h_configuration = replicateGFG65536(gfg); // replicate the gfg 
  for (int n=0; n<gfg->n_atoms; n++) // and cross-parameterize 
  {
    h_configuration->atom[n].sigma = pow(0.5f * (float)(pow(sigma, 6) + pow(h_configuration->atom[n].sigma, 6)), 0.1666666f);
    h_configuration->atom[n].epsilon = sqrt(epsilon * h_configuration->atom[n].epsilon);
  }

  // then do the calc
  // (x,y,z) is (blockx, gridx, gridy)... chunking to 4 parts in x, then will combine results
  int chunk_size = 256, chunks = 4;
  dim3 dimGrid(1024, 1024);
  dim3 dimBlock(chunk_size, 1, 1);

  hipError_t err;
  /* allocate for energy array and configuration on device */
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_repulsion_chunk, sizeof(ftw_Chunk)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_configuration, sizeof(ftw_GFG65536)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy( d_configuration, h_configuration, sizeof(ftw_GFG65536), hipMemcpyHostToDevice ));

  ftw_EnergyArray1024 *h_repulsion = (ftw_EnergyArray1024 *)malloc(sizeof(ftw_EnergyArray1024)); // host structure, for result

  for (int chunk=0; chunk < chunks; chunk++)
  {
    EnergyKernel1024_612<<< dimGrid, dimBlock >>>(d_configuration, NULL, d_repulsion_chunk, NULL, chunk, chunk_size);
    hipDeviceSynchronize(); // block until the device has completed
    err = hipGetLastError();
    if (err != hipSuccess) {printf("CUDA error:  %s\n", hipGetErrorString(err)); exit(1);}
    // retrieve result
    ftw_EnergyArray1024* h_address = (ftw_EnergyArray1024*)((long)h_repulsion + (long)(sizeof(ftw_Chunk) * chunk));
    for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy(h_address, d_repulsion_chunk, sizeof(ftw_Chunk), hipMemcpyDeviceToHost ));
  }

  // free device memory
  hipFree(d_configuration);
  hipFree(d_repulsion_chunk);

  free(h_configuration); // free host memory for replicated configuration
  return h_repulsion;
}

