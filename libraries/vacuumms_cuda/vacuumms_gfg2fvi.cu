#include "hip/hip_runtime.h"
/* vacuumms_gfg2fvi.cu */

//  IN:    A pointer to a ***non-replicated*** polymer configuration as *vacuumms_GFG65536.  
//  OUT:   A free volume intensity is returned as *vacuumms_FVI256.  
//  Input configuration is not modified.  
//  Cross-interaction values are stored for the replicated config only.

#include <vacuumms/gfg2fvi.h>
#include <vacuumms/config_parser.h>
#include <vacuumms/types.h>

#include <stdlib.h>
#include <math.h>

// I took the kernel prototype out of the header file, because the header is included by C/C++ compilers that don't know what a kernel is...
// NOTE:  this uses COMPASS / LJ 6-9 potential
__global__ void EnergyKernel256(vacuumms_GFG65536 *d_configuration, vacuumms_EnergyArray256 *d_attraction, vacuumms_EnergyArray256 *d_repulsion, vacuumms_EnergyArray256 *d_total) 
{
  unsigned int idx = blockIdx.x;
  unsigned int idy = blockIdx.y;
  unsigned int idz = threadIdx.x;

  float repulsion=0;
  float attraction=0;
  float alpha;
  float dx, dy, dz, d, dd;
  float f_resolution_x = d_configuration->box_x / 256;
  float f_resolution_y = d_configuration->box_y / 256;
  float f_resolution_z = d_configuration->box_z / 256;

  float cuda_x = idx * f_resolution_x;
  float cuda_y = idy * f_resolution_y;
  float cuda_z = idz * f_resolution_z;

  // evaluate energy at (cuda_x, cuda_y, cuda_z);
  for (int i=0; i< d_configuration->n_atoms; i++) {
    // central atom
    dx = d_configuration->atom[i].x - cuda_x;
    dy = d_configuration->atom[i].y - cuda_y;
    dz = d_configuration->atom[i].z - cuda_z;
    dd = dx*dx + dy*dy + dz*dz; d = sqrt(dd);
    alpha = pow(d_configuration->atom[i].sigma, 3) / (d * dd);   
    repulsion += d_configuration->atom[i].epsilon * alpha * alpha * alpha;
    attraction += d_configuration->atom[i].epsilon * alpha * alpha;
  } 

  // If NULL pointers are passed for the attraction or repulsion, no values are returned.
  if (d_attraction) d_attraction->energy[idx][idy][idz] = 3 * attraction;
  if (d_repulsion) d_repulsion->energy[idx][idy][idz] = 2 * repulsion;
  if (d_total) d_total->energy[idx][idy][idz] = 2 * repulsion - 3 * attraction;
}

__global__ void EnergyKernel256_612(vacuumms_GFG65536 *d_configuration, vacuumms_EnergyArray256 *d_attraction, vacuumms_EnergyArray256 *d_repulsion, vacuumms_EnergyArray256 *d_total) 
{
  unsigned int idx = blockIdx.x;
  unsigned int idy = blockIdx.y;
  unsigned int idz = threadIdx.x;

  float repulsion=0;
  float attraction=0;
  float sigma_over_r_sq;
  float dx, dy, dz, dd;
  float f_resolution_x = d_configuration->box_x / 256;
  float f_resolution_y = d_configuration->box_y / 256;
  float f_resolution_z = d_configuration->box_z / 256;

  float cuda_x = idx * f_resolution_x;
  float cuda_y = idy * f_resolution_y;
  float cuda_z = idz * f_resolution_z;

  // evaluate energy at (cuda_x, cuda_y, cuda_z);
  for (int i=0; i< d_configuration->n_atoms; i++) {
    // central atom
    dx = d_configuration->atom[i].x - cuda_x;
    dy = d_configuration->atom[i].y - cuda_y;
    dz = d_configuration->atom[i].z - cuda_z;
    dd = dx*dx + dy*dy + dz*dz; 
    sigma_over_r_sq = d_configuration->atom[i].sigma * d_configuration->atom[i].sigma / dd; // squared   
    repulsion += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
    attraction += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
  } 

  // If NULL pointers are passed for the attraction or repulsion, no values are returned.
  if (d_attraction) d_attraction->energy[idx][idy][idz] = 4 * attraction;
  if (d_repulsion) d_repulsion->energy[idx][idy][idz] = 4 * repulsion;
  if (d_total) d_total->energy[idx][idy][idz] = 4 * repulsion - 4 * attraction;
}

__global__ void EnergyKernel512_612(vacuumms_GFG65536 *d_configuration, vacuumms_EnergyArray512 *d_attraction, vacuumms_EnergyArray512 *d_repulsion, vacuumms_EnergyArray512 *d_total) 
{
  unsigned int idx = blockIdx.x;
  unsigned int idy = blockIdx.y;
  unsigned int idz = threadIdx.x;

  float repulsion=0;
  float attraction=0;
  float sigma_over_r_sq;
  float dx, dy, dz, dd;
  float f_resolution_x = d_configuration->box_x / 512;
  float f_resolution_y = d_configuration->box_y / 512;
  float f_resolution_z = d_configuration->box_z / 512;

  float cuda_x = idx * f_resolution_x;
  float cuda_y = idy * f_resolution_y;
  float cuda_z = idz * f_resolution_z;

  // evaluate energy at (cuda_x, cuda_y, cuda_z);
  for (int i=0; i< d_configuration->n_atoms; i++) {
    // central atom
    dx = d_configuration->atom[i].x - cuda_x;
    dy = d_configuration->atom[i].y - cuda_y;
    dz = d_configuration->atom[i].z - cuda_z;
    dd = dx*dx + dy*dy + dz*dz; 
    sigma_over_r_sq = d_configuration->atom[i].sigma * d_configuration->atom[i].sigma / dd; // squared   
    repulsion += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
    attraction += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
  } 

  // If NULL pointers are passed for the attraction or repulsion, no values are returned.
  if (d_attraction) d_attraction->energy[idx][idy][idz] = 4 * attraction;
  if (d_repulsion) d_repulsion->energy[idx][idy][idz] = 4 * repulsion;
  if (d_total) d_total->energy[idx][idy][idz] = 4 * repulsion - 4 * attraction;
}

//  This is the routine to call from outside the library
extern "C" vacuumms_FVI256 *GFGToFVI256(vacuumms_GFG65536 *gfg, float sigma, float epsilon) 
{
  // call energy array then process each val
  vacuumms_EnergyArray256 *era = GFGToRepulsion256(gfg, sigma, epsilon);
  vacuumms_FVI256 *fvi = (vacuumms_FVI256*)malloc(sizeof(vacuumms_FVI256));

  // now process each value...
  for (int i=0; i<256; i++) for (int j=0; j<256; j++) for (int k=0; k<256; k++)
    fvi->intensity[i][j][k] = exp(era->energy[i][j][k]/-298000); // this is arbitrary... should be clarified
  
  return fvi;
}

// Now the C bindings...

//  This routine to be called from outside the library
extern "C" vacuumms_EnergyArray256 *GFGToRepulsion256_612(vacuumms_GFG65536 *gfg, float sigma, float epsilon)
{
  vacuumms_EnergyArray256 	*d_repulsion;
  vacuumms_GFG65536 		*d_configuration;

  // replicate the gfg
  vacuumms_GFG65536 *h_configuration = replicateGFG65536(gfg); 

// and cross-parameterize use 6-12 rule
  for (int n=0; n<gfg->n_atoms; n++)
  {
    h_configuration->atom[n].sigma = 0.5f * (sigma + h_configuration->atom[n].sigma);
    h_configuration->atom[n].epsilon = sqrt(epsilon * h_configuration->atom[n].epsilon);
  }

  // then do the calc
  hipError_t err;
  /* allocate for energy array and configuration on device */
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_repulsion, sizeof(vacuumms_EnergyArray256)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_configuration, sizeof(vacuumms_GFG65536)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy( d_configuration, h_configuration, sizeof(vacuumms_GFG65536), hipMemcpyHostToDevice ));

  dim3 dimGrid(256, 256);
  dim3 dimBlock(256, 1, 1);

  EnergyKernel256_612<<< dimGrid, dimBlock >>>(d_configuration, NULL, d_repulsion, NULL);
  hipDeviceSynchronize(); // block until the device has completed
  err = hipGetLastError();
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err)); 

  // retrieve result
  vacuumms_EnergyArray256 *h_repulsion = (vacuumms_EnergyArray256 *)malloc(sizeof(vacuumms_EnergyArray256));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy(h_repulsion, d_repulsion, sizeof(vacuumms_EnergyArray256), hipMemcpyDeviceToHost ));

  // free device memory
  hipFree(d_configuration);
  hipFree(d_repulsion);

  free(h_configuration); // free host memory for replicated configuration

  return h_repulsion;
}

//  This routine to be called from outside the library
extern "C" vacuumms_EnergyArray512 *GFGToRepulsion512_612(vacuumms_GFG65536 *gfg, float sigma, float epsilon)
{
  vacuumms_EnergyArray512 	*d_repulsion;
  vacuumms_GFG65536 		*d_configuration;


  // replicate the gfg
  vacuumms_GFG65536 *h_configuration = replicateGFG65536(gfg); 

// and cross-parameterize use 6-12 rule
  for (int n=0; n<gfg->n_atoms; n++)
  {
    h_configuration->atom[n].sigma = 0.5f * (sigma + h_configuration->atom[n].sigma);
    h_configuration->atom[n].epsilon = sqrt(epsilon * h_configuration->atom[n].epsilon);
  }

  // then do the calc
  hipError_t err;
  /* allocate for energy array and configuration on device */
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_repulsion, sizeof(vacuumms_EnergyArray512)));
fprintf(stderr, "malloc-ing enrgyarray...\n");
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_configuration, sizeof(vacuumms_GFG65536)));
fprintf(stderr, "malloc-ing gfg ...\n");
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy( d_configuration, h_configuration, sizeof(vacuumms_GFG65536), hipMemcpyHostToDevice ));
fprintf(stderr, "mem copying...\n");

  dim3 dimGrid(512, 512);
  dim3 dimBlock(512, 1, 1);

fprintf(stderr, "launching kernel...\n");
  EnergyKernel512_612<<< dimGrid, dimBlock >>>(d_configuration, NULL, d_repulsion, NULL);
fprintf(stderr, "synchronizing...\n");
  hipDeviceSynchronize(); // block until the device has completed
  err = hipGetLastError();
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err)); 

  // retrieve result
fprintf(stderr, "retrieving result...\n");
  vacuumms_EnergyArray512 *h_repulsion = (vacuumms_EnergyArray512 *)malloc(sizeof(vacuumms_EnergyArray512));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy(h_repulsion, d_repulsion, sizeof(vacuumms_EnergyArray512), hipMemcpyDeviceToHost ));

  // free device memory
  hipFree(d_configuration);
  hipFree(d_repulsion);

  free(h_configuration); // free host memory for replicated configuration

  return h_repulsion;
}

//  This routine to be called from outside the library
extern "C" vacuumms_EnergyArray256 *GFGToRepulsion256(vacuumms_GFG65536 *gfg, float sigma, float epsilon)
{
  vacuumms_EnergyArray256 	*d_repulsion;
  vacuumms_GFG65536 		*d_configuration;

  // replicate the gfg
  vacuumms_GFG65536 *h_configuration = replicateGFG65536(gfg); 

  // and cross-parameterize 
  for (int n=0; n<gfg->n_atoms; n++)
  {
    h_configuration->atom[n].sigma = pow(0.5f * (float)(pow(sigma, 6) + pow(h_configuration->atom[n].sigma, 6)), 0.1666666f);
    h_configuration->atom[n].epsilon = sqrt(epsilon * h_configuration->atom[n].epsilon);
  }

  // then do the calc
  hipError_t err;
  /* allocate for energy array and configuration on device */
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_repulsion, sizeof(vacuumms_EnergyArray256)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_configuration, sizeof(vacuumms_GFG65536)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy( d_configuration, h_configuration, sizeof(vacuumms_GFG65536), hipMemcpyHostToDevice ));

  dim3 dimGrid(256, 256);
  dim3 dimBlock(256, 1, 1);

  EnergyKernel256<<< dimGrid, dimBlock >>>(d_configuration, NULL, d_repulsion, NULL);
  hipDeviceSynchronize(); // block until the device has completed
  err = hipGetLastError();
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err)); 

  // retrieve result
  vacuumms_EnergyArray256 *h_repulsion = (vacuumms_EnergyArray256 *)malloc(sizeof(vacuumms_EnergyArray256));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy(h_repulsion, d_repulsion, sizeof(vacuumms_EnergyArray256), hipMemcpyDeviceToHost ));

  // free device memory
  hipFree(d_configuration);
  hipFree(d_repulsion);

  free(h_configuration); // free host memory for replicated configuration

  return h_repulsion;
}

//  This routine to be called from outside the library
extern "C" vacuumms_EnergyArray256 *GFGToEnergyArray256(vacuumms_GFG65536 *gfg, float sigma, float epsilon)
{
  vacuumms_EnergyArray256 	*d_energy_array;
  vacuumms_GFG65536 		*d_configuration;

  // replicate the gfg
  vacuumms_GFG65536 *h_configuration = replicateGFG65536(gfg); 

  // and cross-parameterize 
  for (int n=0; n<gfg->n_atoms; n++)
  {
    h_configuration->atom[n].sigma = pow(0.5f * (float)(pow(sigma, 6) + pow(h_configuration->atom[n].sigma, 6)), 0.1666666f);
    h_configuration->atom[n].epsilon = sqrt(epsilon * h_configuration->atom[n].epsilon);
  }

  // then do the calc
  hipError_t err;
  /* allocate for energy array and configuration on device */
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_energy_array, sizeof(vacuumms_EnergyArray256)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_configuration, sizeof(vacuumms_GFG65536)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy( d_configuration, h_configuration, sizeof(vacuumms_GFG65536), hipMemcpyHostToDevice ));

  dim3 dimGrid(256, 256);
  dim3 dimBlock(256, 1, 1);

  EnergyKernel256<<< dimGrid, dimBlock >>>(d_configuration, NULL, NULL, d_energy_array);
  hipDeviceSynchronize(); // block until the device has completed
  err = hipGetLastError();
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err)); 

  // retrieve result
  vacuumms_EnergyArray256 *h_energy_array = (vacuumms_EnergyArray256 *)malloc(sizeof(vacuumms_EnergyArray256));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy(h_energy_array, d_energy_array, sizeof(vacuumms_EnergyArray256), hipMemcpyDeviceToHost ));

  // free device memory
  hipFree(d_configuration);
  hipFree(d_energy_array);

  free(h_configuration); // free host memory for replicated configuration

  return h_energy_array;
}

// This is for a traditional LJ 6-12 interaction.  Note that sigma is the value where energy is zero, not the well-bottom as for COMPASS...
// This operates on 'chunks' in x-direction because the domain is too large for the GPU memory
__global__ void EnergyKernel1024_612(	vacuumms_GFG65536 *d_configuration, 
 					vacuumms_Chunk *d_attraction, 
 					vacuumms_Chunk *d_repulsion, 
 					vacuumms_Chunk *d_total, 
 					int chunk, int chunk_size) {
  unsigned int idx = threadIdx.x;
  unsigned int idy = blockIdx.x;
  unsigned int idz = blockIdx.y;

  float repulsion=0;
  float attraction=0;
  float sigma_over_r_sq;
  float dx, dy, dz, dd;
  float f_resolution_x = d_configuration->box_x / 1024;
  float f_resolution_y = d_configuration->box_y / 1024;
  float f_resolution_z = d_configuration->box_z / 1024;

  float cuda_x = (chunk * chunk_size + idx ) * f_resolution_x;
  float cuda_y = idy * f_resolution_y;
  float cuda_z = idz * f_resolution_z;

  // evaluate energy at (cuda_x, cuda_y, cuda_z);
  for (int i=0; i< d_configuration->n_atoms; i++) {
    // central atom
    dx = d_configuration->atom[i].x - cuda_x;
    dy = d_configuration->atom[i].y - cuda_y;
    dz = d_configuration->atom[i].z - cuda_z;
    dd = dx*dx + dy*dy + dz*dz;
    sigma_over_r_sq = d_configuration->atom[i].sigma * d_configuration->atom[i].sigma / dd;
    repulsion  += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
    attraction += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
  } 

  // If NULL pointers are passed, then no values are recorded.
  if (d_attraction) d_attraction->energy[idx][idy][idz] = 4 * attraction;
  if (d_repulsion) d_repulsion->energy[idx][idy][idz] = 4 * repulsion;
  if (d_total) d_total->energy[idx][idy][idz] = 4 * repulsion - 4 * attraction;
}

// This routine to be called from outside the library
extern "C" vacuumms_EnergyArray1024 *GFGToEnergyArray1024_612(vacuumms_GFG65536 *gfg, float sigma, float epsilon)
{
  vacuumms_Chunk 		*d_energy_array_chunk;
  vacuumms_GFG65536 		*d_configuration;

  vacuumms_GFG65536 *h_configuration = replicateGFG65536(gfg); // replicate the gfg 
  for (int n=0; n<gfg->n_atoms; n++) // and cross-parameterize 
  {
    h_configuration->atom[n].sigma = pow(0.5f * (float)(pow(sigma, 6) + pow(h_configuration->atom[n].sigma, 6)), 0.1666666f);
    h_configuration->atom[n].epsilon = sqrt(epsilon * h_configuration->atom[n].epsilon);
  }

  // then do the calc
  // (x,y,z) is (blockx, gridx, gridy)... chunking to 4 parts in x, then will combine results
  int chunk_size = 256, chunks = 4;
  dim3 dimGrid(1024, 1024);
  dim3 dimBlock(chunk_size, 1, 1);

  hipError_t err;
  /* allocate for energy array and configuration on device */
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_energy_array_chunk, sizeof(vacuumms_Chunk)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_configuration, sizeof(vacuumms_GFG65536)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy( d_configuration, h_configuration, sizeof(vacuumms_GFG65536), hipMemcpyHostToDevice ));

  vacuumms_EnergyArray1024 *h_energy_array = (vacuumms_EnergyArray1024 *)malloc(sizeof(vacuumms_EnergyArray1024)); // host structure, for result

  for (int chunk=0; chunk < chunks; chunk++)
  {
    EnergyKernel1024_612<<< dimGrid, dimBlock >>>(d_configuration, NULL, NULL, d_energy_array_chunk, chunk, chunk_size);
    hipDeviceSynchronize(); // block until the device has completed
    err = hipGetLastError();
    if (err != hipSuccess) {printf("CUDA error:  %s\n", hipGetErrorString(err)); exit(1);}
    // retrieve result
    vacuumms_EnergyArray1024* h_address = (vacuumms_EnergyArray1024*)((long)h_energy_array + (long)(sizeof(vacuumms_Chunk) * chunk));
    for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy(h_address, d_energy_array_chunk, sizeof(vacuumms_Chunk), hipMemcpyDeviceToHost ));
  }

  // free device memory
  hipFree(d_configuration);
  hipFree(d_energy_array_chunk);

  free(h_configuration); // free host memory for replicated configuration
  return h_energy_array;
}

// This routine to be called from outside the library
extern "C" vacuumms_EnergyArray1024 *GFGToRepulsion1024_612(vacuumms_GFG65536 *gfg, float sigma, float epsilon)
{
  vacuumms_Chunk 		*d_repulsion_chunk;
  vacuumms_GFG65536 		*d_configuration;

  vacuumms_GFG65536 *h_configuration = replicateGFG65536(gfg); // replicate the gfg 
  for (int n=0; n<gfg->n_atoms; n++) // and cross-parameterize 
  {
    h_configuration->atom[n].sigma = pow(0.5f * (float)(pow(sigma, 6) + pow(h_configuration->atom[n].sigma, 6)), 0.1666666f);
    h_configuration->atom[n].epsilon = sqrt(epsilon * h_configuration->atom[n].epsilon);
  }

  // then do the calc
  // (x,y,z) is (blockx, gridx, gridy)... chunking to 4 parts in x, then will combine results
  int chunk_size = 256, chunks = 4;
  dim3 dimGrid(1024, 1024);
  dim3 dimBlock(chunk_size, 1, 1);

  hipError_t err;
  /* allocate for energy array and configuration on device */
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_repulsion_chunk, sizeof(vacuumms_Chunk)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_configuration, sizeof(vacuumms_GFG65536)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy( d_configuration, h_configuration, sizeof(vacuumms_GFG65536), hipMemcpyHostToDevice ));

  vacuumms_EnergyArray1024 *h_repulsion = (vacuumms_EnergyArray1024 *)malloc(sizeof(vacuumms_EnergyArray1024)); // host structure, for result

  for (int chunk=0; chunk < chunks; chunk++)
  {
    EnergyKernel1024_612<<< dimGrid, dimBlock >>>(d_configuration, NULL, d_repulsion_chunk, NULL, chunk, chunk_size);
    hipDeviceSynchronize(); // block until the device has completed
    err = hipGetLastError();
    if (err != hipSuccess) {printf("CUDA error:  %s\n", hipGetErrorString(err)); exit(1);}
    // retrieve result
    vacuumms_EnergyArray1024* h_address = (vacuumms_EnergyArray1024*)((long)h_repulsion + (long)(sizeof(vacuumms_Chunk) * chunk));
    for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy(h_address, d_repulsion_chunk, sizeof(vacuumms_Chunk), hipMemcpyDeviceToHost ));
  }

  // free device memory
  hipFree(d_configuration);
  hipFree(d_repulsion_chunk);

  free(h_configuration); // free host memory for replicated configuration
  return h_repulsion;
}

